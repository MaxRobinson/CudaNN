
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
// #include <cuda.h>
#include "hipblas.h"

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>



#define DEBUG false

#define MAX 100
#define MULTIPLIER 1.0

#define MAX_ARRAY_SIZE 1<<20

#define index(i,j,ld) (((j)*(ld))+(i))

// Struct to hold multiple timing metrics per run for comparison
struct MultTiming {
    float initTime;
    float overallTime;
};

// Print Matrix on host
void printMat(float*P,int uWP,int uHP){
    //printf("\n %f",P[1]);
    int i,j;
    for(i=0;i<uHP;i++){
        printf("\n");
        for(j=0;j<uWP;j++)
            printf("%f ",P[index(i,j,uHP)]);
    }
    printf("\n");
}

// For printing entire network from device
// meant for debugging.
void printNetwork(float* dev_input, float* dev_w1, float* dev_w2, float* dev_w3,
    int input_layer_size, int hidden_layer_1_size, int hidden_layer_2_size, int output_layer_size){
    
    float *h_input = (float*)malloc(1*input_layer_size*sizeof(float));
    hipblasGetMatrix(1, input_layer_size, sizeof(float), dev_input, 1, h_input, 1);
    
    float *h_w1 = (float*)malloc(input_layer_size*hidden_layer_1_size*sizeof(float));
    hipblasGetMatrix(input_layer_size, hidden_layer_1_size, sizeof(float), dev_w1, input_layer_size, h_w1, input_layer_size);

    float *h_w2 = (float*)malloc(hidden_layer_1_size*hidden_layer_2_size*sizeof(float));
    hipblasGetMatrix(hidden_layer_1_size, hidden_layer_2_size, sizeof(float), dev_w2, hidden_layer_1_size, h_w2, hidden_layer_1_size);
    
    float *h_w3 = (float*)malloc(hidden_layer_2_size*output_layer_size*sizeof(float));
    hipblasGetMatrix(hidden_layer_2_size, output_layer_size, sizeof(float), dev_w3, hidden_layer_2_size, h_w3, hidden_layer_2_size);


    printMat(h_input, input_layer_size, 1);
    printMat(h_w1, hidden_layer_1_size, input_layer_size);
    printMat(h_w2, hidden_layer_2_size, hidden_layer_1_size);
    printMat(h_w3, output_layer_size, hidden_layer_2_size); 
}

/** 
* from: https://devtalk.nvidia.com/default/topic/524307/need-help-with-kernel-execution-parameters/
* used to wrap cuda calls with error possiblility. 
*/ 
__host__ int cudaCall(hipError_t value, int line) {                                                                                      
    hipError_t _m_cudaStat = value;                                                                                
    if (_m_cudaStat != hipSuccess) {                                                                               
            printf("Error %s at line %d \n", hipGetErrorString(_m_cudaStat), line);           
            exit(1);                                                                                                                        
    } 
    return 0;
}
#define CUDA_CALL(value) cudaCall( value, __LINE__)

/**
* modified from https://docs.nvidia.com/cuda/curand/host-api-overview.html#host-api-overview 
* used to wrap cuRAND calls with error possilbility
*/
__host__ int cuRandCall(hiprandStatus_t value, const char* file, int line){ 
    if( value != HIPRAND_STATUS_SUCCESS) {
        printf("Error at %s:%d\n",__FILE__,__LINE__);
        return EXIT_FAILURE;
    }
    return 0;
}
#define CURAND_CALL(value) cuRandCall(value, __FILE__, __LINE__)


// apply sigmoid function to a value of arrays
// sigmoid = (1 / (1 + e^(-input)))
__global__ void sigmoid(float* input, int num_elements){
    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x; 
    if(tid < num_elements)
	{
        float value = 1.0 / (1.0 + exp(-1*input[tid]));
        input[tid] = value;
    }
}

/**
* Creates a CUDA event at the current time
* Provided by grader
*
* @param None
*
* @return time The cuda event for the current time
*/
__host__
hipEvent_t getTime(hipStream_t stream)
{
    hipEvent_t time;

    hipEventCreate(&time);
    hipEventRecord(time, stream);

    return time;
}

/**
* Helper function to init arrays on host
*
*/ 
__host__ void init_array(int ** array, int arraySize, int offset){
    int* array_actual = *array; 
    for(int i = 0; i < arraySize; i++){
        array_actual[i] = i + offset; 
    }
}

// randomly initialize array
void initWeights(float ** d_array, int arraySize){
    #if DEBUG
    printf("init Weights\n");
    #endif
    // init arrays on the device using cuRAND
    // code adapted from https://docs.nvidia.com/cuda/curand/host-api-overview.html#host-api-overview 
    hiprandGenerator_t gen;
    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
    /* Generate n floats on device */
    /* This code generates floats on the device thus is calling a kernel
        to perform this operation */
    CURAND_CALL(hiprandGenerateUniform(gen, *d_array, arraySize));
}

// multiplies an input vector 1 row x yColumns 
float* layerMult(float* input_values, int input_size, 
                float * weights, int weight_col_size){
    hipblasStatus_t status;
    // const float alpha = 1.0;

    float* layer_outputs;
    status = cublasAlloc((1*weight_col_size), sizeof(float),(void **) &layer_outputs);


    cublasSgemm('n', 'n', 
        1, weight_col_size, input_size, // a_rows, b_columns, a_columns 
        1, // alpha
        input_values, 1, // a, a rows leading dimension (height)
        weights, input_size, // b, rows of weights (= input_size)
        0, // beta 
        layer_outputs, 1); // output,  output leading dimension (height)
    
    
    // perform sigmoid transform to hidden layer values
    // input now has hidden layer 1 values
    // printf("about to run sigmoid\n");
    sigmoid<<<1, weight_col_size>>>(layer_outputs, weight_col_size);
    
    #if DEBUG
    float* c = (float *)malloc (1 * weight_col_size * sizeof (float));
    status = cublasGetMatrix (1, weight_col_size, sizeof(*c), layer_outputs, 1, c, 1);
    if (status != CUBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        exit(1);
    }

    for (int j = 0; j < 1; j++) {
        for (int i = 0; i < weight_col_size; i++) {
            printf("%f ",c[index(i,j,1)]);
        }
        printf ("\n");
    }
    #endif

    return layer_outputs;
}

// Calculates the forward pass of a neural network with 2 hidden layers
// each layer is calculation of 1/(1-exp(-1*sum(x_i*w_i))) for each node in the next layer 
// where x is an input vector and w corresponds to the weights for that input node per each input value from x
float* forwardPass(float* input_values, int input_size,
                float* weights1, int hidden_layer_1_size,
                float* weights2, int hidden_layer_2_size,
                float* weights3, int output_layer_size
            ){
    #if DEBUG
    printf("Forward Pass \n");
    #endif
    
    // wieght matrix size = input_size X hidden_layer_size
    // Weigth matrix is stored as each array of weights is a column. 
    // So the height of W (number of rows) is = to the number of nodes in the previous layer
    // The width of W (number of columns) is = to the number of nodes in the next layer
    
    // allocate the array to store the output of the input x hidden layer weigths = hidden_layer 1 values
    float* layer1_outputs = layerMult(input_values, input_size, weights1, hidden_layer_1_size);
    float* layer2_outputs = layerMult(layer1_outputs, hidden_layer_1_size, weights2, hidden_layer_2_size);
    float* output = layerMult(layer2_outputs, hidden_layer_2_size, weights3, output_layer_size);

    return output;
}

/**
* Main program
*
*/
int main(int argc, char** argv) {
    hipblasStatus_t status;
    cublasInit();
    int input_layer_size = 3; 
    int hidden_layer_1_size = 10; 
    int hidden_layer_2_size = 5;
    int output_layer_size = 3; 

    // wieght matrix size = input_size X hidden_layer_size
    // Weigth matrix is stored as each array of weights is a column. 
    // So the hieght of W (number of rows) is = to the number of inputs into the next layer's node
    // The width of W (number of columns) is = to the number of nodes in the next layer
    float * input_values;
    float* weights1; 
    float* weights2;
    float* weights3; 
    
    // cuda malloc input value space on GPU
    CUDA_CALL(hipMalloc((void **) &input_values, (input_layer_size) * sizeof(float)));

    // // cuda malloc space for weight matrices on GPU
    // CUDA_CALL(cudaMalloc((void **) &weights1, (input_layer_size * hidden_layer_1_size) * sizeof(float)));
    // CUDA_CALL(cudaMalloc((void **) &weights2, (hidden_layer_1_size * hidden_layer_2_size) * sizeof(float)));
    // CUDA_CALL(cudaMalloc((void **) &weights3, (hidden_layer_2_size * output_layer_size) * sizeof(float)));

    status = cublasAlloc((input_layer_size * hidden_layer_1_size), sizeof(float), (void **) &weights1);
    status = cublasAlloc((hidden_layer_1_size * hidden_layer_2_size), sizeof(float), (void **) &weights2);
    status = cublasAlloc((hidden_layer_2_size * output_layer_size), sizeof(float), (void **) &weights3);

    // // init input as random for testing for now
    initWeights(&input_values, input_layer_size);
    initWeights(&weights1, input_layer_size * hidden_layer_1_size);
    initWeights(&weights2, hidden_layer_1_size * hidden_layer_2_size);
    initWeights(&weights3, hidden_layer_2_size * output_layer_size);

    // printMat(C, input_layer_size, 1);
    #if DEBUG
    printNetwork(input_values, weights1, weights2, weights3, 
                input_layer_size, hidden_layer_1_size, hidden_layer_2_size, output_layer_size);
    #endif

    // output is still on device
    float* dev_output = forwardPass(input_values, input_layer_size,
        weights1, hidden_layer_1_size,
        weights2, hidden_layer_2_size,
        weights3, output_layer_size
    );

    float* h_output = (float *)malloc (1 * output_layer_size * sizeof (float));
    status = hipblasGetMatrix (1, output_layer_size, sizeof(*h_output), dev_output, 1, h_output, 1);

    printf("Network output: ");
    printMat(h_output, output_layer_size, 1);
    return true;
}

